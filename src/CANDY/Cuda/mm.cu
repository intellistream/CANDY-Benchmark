#include "hip/hip_runtime.h"
//
// Created by tony on 12/06/24.
//
#include <torch/torch.h>
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
// Utility function to check CUDA errors
inline void checkCudaError(hipError_t result, char const *const func, const char *const file, int64_t const line) {
  if (result != hipSuccess) {
    std::cerr << "CUDA error = " << static_cast<int64_t>(result) << " at " <<
              file << ":" << line << " '" << func << "' \n" << hipGetErrorString(result) << std::endl;
    exit(1);
  }
}

#define CHECK_CUDA_ERROR(val) checkCudaError((val), #val, __FILE__, __LINE__)
__global__ void matrixMulCUDA(float *a, float *b, float *c, int64_t M, int64_t N, int64_t K) {
  int64_t row = blockIdx.y * blockDim.y + threadIdx.y;
  int64_t col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < M && col < K) {
    float sum = 0.0;
    for (int64_t i = 0; i < N; i++) {
      sum += a[row * N + i] * b[i * K + col];
    }
    c[row * K + col] = sum;
  }
}

torch::Tensor CudaMM(torch::Tensor a, torch::Tensor b) {
  // Ensure input tensors are on the GPU and are contiguous
  a = a.to(at::kCUDA).contiguous();
  b = b.to(at::kCUDA).contiguous();

  // Dimensions
  int64_t M = a.size(0);
  int64_t N = a.size(1);
  int64_t K = b.size(1);

  // Create the output tensor on the GPU

  torch::Tensor c = torch::zeros({M, K}).to(at::kCUDA);

  // Define block and grid sizes
  dim3 threadsPerBlock(4, 4);
  dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                     (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

  // Launch the kernel
  matrixMulCUDA<<<blocksPerGrid, threadsPerBlock>>>(a.data_ptr<float>(),
                                                    b.data_ptr<float>(),
                                                    c.data_ptr<float>(),
                                                    M,
                                                    N,
                                                    K);

  // Wait for GPU to finish before accessing on host
  CHECK_CUDA_ERROR(hipDeviceSynchronize());

  // Transfer the result tensor to the CPU
  return c.to(torch::kCPU);
}